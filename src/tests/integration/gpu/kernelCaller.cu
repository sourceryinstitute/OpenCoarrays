#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

extern "C" void cudaPrint(int *data, int n, int me, char *name);
extern "C" void cudaAdd(int* data_in1, int* data_in2, int *data_out, int n);
extern "C" void cudaDot(float* in1, float* in2, float* out, int n);

#define MAX_BLOCK_SZ 1024

__global__ void printOnCuda(int *data, int n)
{
  int i = 0;
  int me = blockIdx.x*blockDim.x + threadIdx.x;
  if (me == 0)
    for(i=0;i<n;i++)
      printf("From CUDA data[%d] = %d\n",i,data[i]);
}

__global__ void assignOnCuda(int *data, int n)
{
  int i = 0;
  int me = blockIdx.x*blockDim.x + threadIdx.x;
  if (me == 0)
    for(i=0;i<n;i++)
      data[i] = me + i;
//      printf("From CUDA data[%d] = %d\n",i,data[i]);
}

__global__ void Dev_dot(float x[], float y[], float z[], long n) {
   /* Use tmp to store products of vector components in each block */
   /* Can't use variable dimension here                            */
   __shared__ float tmp[MAX_BLOCK_SZ];
   long t = blockDim.x * blockIdx.x + threadIdx.x;
   int loc_t = threadIdx.x;

   if (t < n) tmp[loc_t] = x[t]*y[t];
   __syncthreads();

   /* This uses a tree structure to do the addtions */
   for (int stride = blockDim.x/2; stride >  0; stride /= 2) {
      if (loc_t < stride)
         tmp[loc_t] += tmp[loc_t + stride];
      __syncthreads();
   }

   /* Store the result from this cache block in z[blockIdx.x] */
   if (threadIdx.x == 0) {
      z[blockIdx.x] = tmp[0];
   }
}  /* Dev_dot */

__global__ void add_cuda_int(int* data_in1, int* data_in2, int *data_out, int n)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i < n)
    data_out[i] = data_in1[i] + data_in2[i];

  __syncthreads();
}

void cudaAssign(int *data, int n, char *name)
{
  printf("Processor %s writes data on Cuda\n",name);
  assignOnCuda<<<1,32>>>(data,n);
}
extern "C"
{
void cudaDot(float *in1, float *in2, float *out, int n)
{
  float *partial_dot;
  int nThreads = 1024, i=0;
  int nBlocks = ((n-1)/nThreads)+1;
  float *d_in1,*d_in2;
//  cublasHandle_t handle;

  hipMallocManaged(&partial_dot, nBlocks * sizeof(float));
  hipDeviceSynchronize();

  *out = 0.0;

  hipHostGetDevicePointer((void **) &d_in1, (void *) in1, 0);
  hipHostGetDevicePointer((void **) &d_in2, (void *) in2, 0);

  Dev_dot<<<nBlocks,nThreads>>>(d_in1,d_in2,partial_dot,n);
//  cublasCreate (&handle);
//  cublasSdot(handle,n,d_in1,1,d_in2,1,out);
  hipDeviceSynchronize();

  for(i=0;i<nBlocks;i++)
    {
      *out += partial_dot[i];
    }

  hipFree(partial_dot);

}

void manual_mapped_cudaDot(float* in1, float* in2, float *out, int n, int img)
{
  float *partial_dot;
  float *in1_d, *in2_d;
  int nThreads = 64, i=0;
  int nBlocks = ((n-1)/nThreads)+1;
  int count = 0;

  hipGetDeviceCount(&count);
  hipSetDevice(img%count);

  hipMallocManaged(&partial_dot, nBlocks * sizeof(float));
  *out = 0.0;

  hipHostRegister(in1,n*sizeof(float),hipHostRegisterMapped);
  hipHostRegister(in2,n*sizeof(float),hipHostRegisterMapped);

  hipHostGetDevicePointer(&in1_d,in1,0);
  hipHostGetDevicePointer(&in2_d,in2,0);

  Dev_dot<<<nBlocks,nThreads>>>(in1_d,in2_d,partial_dot,n);
  hipDeviceSynchronize();

  for(i=0;i<nBlocks;i++)
    *out += partial_dot[i];

  hipFree(partial_dot);
}


void manual_cudaDot(float* in1, float* in2, float *out, int n, int img)
{
  float *partial_dot;
  float *in1_d, *in2_d;
  int nThreads = 64, i=0;
  int nBlocks = ((n-1)/nThreads)+1;
  int count = 0;

  hipGetDeviceCount(&count);
  hipSetDevice(img%count);

  hipMallocManaged(&partial_dot, nBlocks * sizeof(float));
  *out = 0.0;

  hipMalloc(&in1_d,sizeof(float)*n);
  hipMalloc(&in2_d,sizeof(float)*n);

  hipMemcpy(in1_d,in1,n*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(in2_d,in2,n*sizeof(float),hipMemcpyHostToDevice);

  Dev_dot<<<nBlocks,nThreads>>>(in1_d,in2_d,partial_dot,n);
  hipDeviceSynchronize();

  for(i=0;i<nBlocks;i++)
    *out += partial_dot[i];

  hipFree(partial_dot);
  hipFree(in1_d);
  hipFree(in2_d);
}

void cudaAdd(int *data_in1, int *data_in2, int *data_out, int n)
{
  int nThreads = 64;
  int nBlocks = ((n-1)/nThreads)+1;
//  printf("n: %d\n",n);
//  cudaDeviceSynchronize();
  add_cuda_int<<<nBlocks,nThreads>>>(data_in1,data_in2,data_out,n);
  hipDeviceSynchronize();
}
void cudaPrint(int *data, int n, int me, char *name)
{
  printf("Image %d on processor %s reads data on Cuda\n",me,name);
  printOnCuda<<<1,32>>>(data,n);
}
}
